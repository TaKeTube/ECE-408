#include "hip/hip_runtime.h"

#include <wb.h>

#define wbCheck(stmt)                                                      \
    do                                                                     \
    {                                                                      \
        hipError_t err = stmt;                                            \
        if (err != hipSuccess)                                            \
        {                                                                  \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                    \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err)); \
            return -1;                                                     \
        }                                                                  \
    } while (0)

#define TILE_WIDTH 32

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns)
{
    //@@ Insert code to implement matrix multiplication here
    //@@ You have to use shared memory for this MP
    __shared__ float subTileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float subTileB[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;
    int mnum = (numAColumns - 1) / TILE_WIDTH + 1;
    float sum = 0;

    for (int m = 0; m < mnum; ++m)
    {
        if ((Row < numARows) && (m * TILE_WIDTH + tx < numAColumns))
        {
            subTileA[ty][tx] = A[Row * numAColumns + m * TILE_WIDTH + tx];
        }
        else
        {
            subTileA[ty][tx] = 0;
        }
        if ((Col < numBColumns) && (m * TILE_WIDTH + ty < numBRows))
        {
            subTileB[ty][tx] = B[(m * TILE_WIDTH + ty) * numBColumns + Col];
        }
        else
        {
            subTileB[ty][tx] = 0;
        }
        __syncthreads();
        for (int k = 0; k < TILE_WIDTH; ++k)
            sum += subTileA[ty][k] * subTileB[k][tx];
        __syncthreads();
    }

    if ((Row < numCRows) && (Col < numCColumns))
        C[Row * numCColumns + Col] = sum;
}

int main(int argc, char **argv)
{
    wbArg_t args;
    float *hostA; // The A matrix
    float *hostB; // The B matrix
    float *hostC; // The output C matrix
    float *deviceA;
    float *deviceB;
    float *deviceC;
    int numARows;    // number of rows in the matrix A
    int numAColumns; // number of columns in the matrix A
    int numBRows;    // number of rows in the matrix B
    int numBColumns; // number of columns in the matrix B
    int numCRows;    // number of rows in the matrix C (you have to set this)
    int numCColumns; // number of columns in the matrix C (you have to set
                     // this)

    args = wbArg_read(argc, argv);

    hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                              &numAColumns);
    hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                              &numBColumns);
    //@@ Set numCRows and numCColumns
    numCRows = numARows;
    numCColumns = numBColumns;
    //@@ Allocate the hostC matrix
    hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));

    //@@ Allocate GPU memory here
    hipMalloc((void **)&deviceA, numARows * numAColumns * sizeof(float));
    hipMalloc((void **)&deviceB, numBRows * numBColumns * sizeof(float));
    hipMalloc((void **)&deviceC, numCRows * numCColumns * sizeof(float));

    //@@ Copy memory to the GPU here
    hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float), hipMemcpyHostToDevice);

    //@@ Initialize the grid and block dimensions here
    dim3 DimGrid(ceil(((float)numCColumns) / TILE_WIDTH), ceil(((float)numCRows) / TILE_WIDTH), 1);
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);

    //@@ Launch the GPU Kernel here
    matrixMultiplyShared<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC,
                                                numARows, numAColumns,
                                                numBRows, numBColumns,
                                                numCRows, numCColumns);

    hipDeviceSynchronize();

    //@@ Copy the GPU memory back to the CPU here
    hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float), hipMemcpyDeviceToHost);

    //@@ Free the GPU memory here
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    wbSolution(args, hostC, numCRows, numCColumns);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}
