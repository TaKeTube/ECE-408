#include "hip/hip_runtime.h"
// MP 1
#include <wb.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len)
{
    //@@ Insert code to implement vector addition here
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < len)
        out[i] = in1[i] + in2[i];
}

int main(int argc, char **argv)
{
    wbArg_t args;
    int inputLength;
    float *hostInput1;
    float *hostInput2;
    float *hostOutput;
    float *deviceInput1;
    float *deviceInput2;
    float *deviceOutput;

    args = wbArg_read(argc, argv);

    hostInput1 =
        (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 =
        (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *)malloc(inputLength * sizeof(float));

    //@@ Allocate GPU memory here
    hipMalloc((void **)&deviceInput1, inputLength * sizeof(float));
    hipMalloc((void **)&deviceInput2, inputLength * sizeof(float));
    hipMalloc((void **)&deviceOutput, inputLength * sizeof(float));

    //@@ Copy memory to the GPU here
    hipMemcpy(deviceInput1, hostInput1, inputLength * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceInput2, hostInput2, inputLength * sizeof(float), hipMemcpyHostToDevice);

    //@@ Initialize the grid and block dimensions here
    dim3 DimGrid(ceil((float)inputLength / 256), 1, 1);
    dim3 DimBlock(256, 1, 1);

    //@@ Launch the GPU Kernel here
    vecAdd<<<DimGrid, DimBlock>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

    hipDeviceSynchronize();

    //@@ Copy the GPU memory back to the CPU here
    hipMemcpy(hostOutput, deviceOutput, inputLength * sizeof(float), hipMemcpyDeviceToHost);

    //@@ Free the GPU memory here
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}
