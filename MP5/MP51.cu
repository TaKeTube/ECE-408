#include "hip/hip_runtime.h"
// MP Reduction
// Given a list (lst) of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];

#include <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt)                                                      \
    do                                                                     \
    {                                                                      \
        hipError_t err = stmt;                                            \
        if (err != hipSuccess)                                            \
        {                                                                  \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                    \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err)); \
            return -1;                                                     \
        }                                                                  \
    } while (0)

__global__ void total(float *input, float *output, int len)
{
    //@@ Load a segment of the input vector into shared memory
    //@@ Traverse the reduction tree
    //@@ Write the computed sum of the block to the output vector at the
    //@@ correct index
    __shared__ float partSum[2 * BLOCK_SIZE];

    unsigned int t = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * blockDim.x;
    unsigned int id1 = start + t;
    unsigned int id2 = start + blockDim.x + t;
    partSum[t] = (id1 < len) ? input[id1] : 0;
    partSum[blockDim.x + t] = (id2 < len) ? input[id2] : 0;

    for (unsigned int stride = blockDim.x; stride >= 1; stride >>= 1)
    {
        __syncthreads();
        if (t < stride)
            partSum[t] += partSum[t + stride];
    }

    if (t == 0)
        output[blockIdx.x] = partSum[0];
}

int main(int argc, char **argv)
{
    int ii;
    wbArg_t args;
    float *hostInput;  // The input 1D list
    float *hostOutput; // The output list
    float *deviceInput;
    float *deviceOutput;
    int numInputElements;  // number of elements in the input list
    int numOutputElements; // number of elements in the output list

    args = wbArg_read(argc, argv);

    hostInput =
        (float *)wbImport(wbArg_getInputFile(args, 0), &numInputElements);

    numOutputElements = numInputElements / (BLOCK_SIZE << 1);
    if (numInputElements % (BLOCK_SIZE << 1))
    {
        numOutputElements++;
    }
    hostOutput = (float *)malloc(numOutputElements * sizeof(float));

    wbLog(TRACE, "The number of input elements in the input is ",
          numInputElements);
    wbLog(TRACE, "The number of output elements in the input is ",
          numOutputElements);

    //@@ Allocate GPU memory here
    hipMalloc((void **)&deviceInput, numInputElements * sizeof(float));
    hipMalloc((void **)&deviceOutput, numOutputElements * sizeof(float));

    //@@ Copy memory to the GPU here
    hipMemcpy(deviceInput, hostInput, numInputElements * sizeof(float), hipMemcpyHostToDevice);

    //@@ Initialize the grid and block dimensions here
    dim3 DimGrid(numOutputElements, 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);

    //@@ Launch the GPU Kernel here
    total<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, numInputElements);

    hipDeviceSynchronize();

    //@@ Copy the GPU memory back to the CPU here
    hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(float), hipMemcpyDeviceToHost);

    /********************************************************************
   * Reduce output vector on the host
   * NOTE: One could also perform the reduction of the output vector
   * recursively and support any size input. For simplicity, we do not
   * require that for this lab.
   ********************************************************************/
    for (ii = 1; ii < numOutputElements; ii++)
    {
        hostOutput[0] += hostOutput[ii];
    }

    //@@ Free the GPU memory here
    hipFree(deviceInput);
    hipFree(deviceOutput);

    wbSolution(args, hostOutput, 1);

    free(hostInput);
    free(hostOutput);

    return 0;
}